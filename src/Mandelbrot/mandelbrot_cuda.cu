/*
  This program is imported from:
  https://gist.github.com/andrejbauer/7919569

  This program is an adaptation of the Mandelbrot program
  from the Programming Rosetta Stone, see
  http://rosettacode.org/wiki/Mandelbrot_set

  See http://www.imagemagick.org/Usage/color_mods/ for what ImageMagick
  can do. It can do a lot.
*/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdint.h>
#include <hip/hip_runtime.h>

__global__ void kernal(int *device_results, int total_pixels, int xres, double xmin, double ymax, double dx, double dy, int maxiter){
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;

    if(threadId >= total_pixels){
        return;
    }

    double x, y; /* Coordinates of the current point in the complex plane. */
    //double u, v; /* Coordinates of the iterated point. */
    /* Pixel counters */

    int i = threadId % xres;
    int j =  threadId / xres;
    int k; /* Iteration counter */
    y = ymax - j * dy;
    double u = 0.0;
    double v= 0.0;
    double u2 = u * u;
    double v2 = v*v;
    x = xmin + i * dx;

    /* iterate the point */
    for (k = 1; k < maxiter && (u2 + v2 < 4.0); k++) {
        v = 2 * u * v + y;
        u = u2 - v2 + x;
        u2 = u * u;
        v2 = v * v;
    };
    device_results[threadId] = k;
}


int main(int argc, char* argv[])
{
    //Check status
    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }
    int *results_host;
    int *results_device;
    /* Parse the command line arguments. */
    if (argc != 2) {
        printf("Usage:   %s <out.ppm>\n", argv[0]);
        printf("Example: %s pic.ppm\n", argv[0]);
        return -1;
    }

    /* The window in the plane. */
    const double xmin = -2;
    const double xmax = 1;
    const double ymin = -1.5;
    const double ymax = 1.5;

    /* Maximum number of iterations, at most 65535. */
    const uint16_t maxiter = 1000;

    /* Image size, width is given, height is computed. */
    const int xres = 1000;
    const int yres = 1000;
    const int pixels = xres * yres;

    /* Precompute pixel width and height. */
    double dx=(xmax-xmin)/xres;
    double dy=(ymax-ymin)/yres;

    const int blockSize = 32;
    int gridCount = pixels / blockSize;


    if (pixels % blockSize != 0){
        gridCount += 1;
    }

    int pixelsPerBlock = pixels / gridCount;
    if (pixels % gridCount != 0){
        pixelsPerBlock += 1;
    }

    dim3 dimGrid(gridCount, 1, 1);
    dim3 dimBlock(blockSize,1, 1);

    int results_bytes = sizeof(int)*pixels;
    results_host = (int*)malloc(results_bytes);
    hipMalloc(&results_device, results_bytes);

    kernal<<<dimGrid,dimBlock>>>(results_device, pixels, xres, xmin, ymax, dx, dy, maxiter);
    hipDeviceSynchronize();

    hipMemcpy(results_host, results_device, results_bytes, hipMemcpyDeviceToHost);


    /* The output file name */
    const char* filename = argv[1];

    /* Open the file and write the header. */
    FILE * fp = fopen(filename,"wb");
//char *comment="# Mandelbrot set";/* comment should start with # */

    /*write ASCII header to the file*/
    fprintf(fp,
            "P6\n# Mandelbrot, xmin=%lf, xmax=%lf, ymin=%lf, ymax=%lf, maxiter=%d\n%d\n%d\n%d\n",
            xmin, xmax, ymin, ymax, maxiter, xres, yres, (maxiter < 256 ? 256 : maxiter));

    /* compute  pixel color and write it to file */
    for(int z = 0; z < pixels; z++){
        int k = results_host[z];
        if (k >= maxiter) {
            /* interior */
            const unsigned char black[] = {0, 0, 0, 0, 0, 0};
            fwrite (black, 6, 1, fp);
        }
        else {
            /* exterior */
            unsigned char color[6];
            color[0] = k >> 8;
            color[1] = k & 255;
            color[2] = k >> 8;
            color[3] = k & 255;
            color[4] = k >> 8;
            color[5] = k & 255;
            fwrite(color, 6, 1, fp);
        };
    }





    fclose(fp);
    return 0;
}
