#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>


#include <omp.h>
#include <hip/hip_runtime.h>


int coord_to_index(int row_coord, int col_coord, int columns){
    return (row_coord * columns) + col_coord;
}


void random_square_matrix(int n, float *matrix_out){

    for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++){
            matrix_out[coord_to_index(i, j, n)] = rand();
        }
    }
}

__global__ void matrix_matrix_multiply(int matrix_dimensions_a[], int matrix_dimensions_b[], float matrix_a[], float matrix_b[], float *output_matrix){
    int i, j, k;


    if(matrix_dimensions_a[1] != matrix_dimensions_b[0]){
        printf("Matrices are incompatible shapes to multiply\n");
        printf("Matrix A has %d columns\n", matrix_dimensions_a[1]);
        printf("Matrix B has %d rows\n", matrix_dimensions_b[0]);
    }

    for(i = 0; i < matrix_dimensions_a[0]; i++){
        for(j = 0; j < matrix_dimensions_b[1]; j++){
            for(k = 0; k < matrix_dimensions_b[0]; k++){
                output_matrix[(i*matrix_dimensions_b[1]) + j] = 0;
            }
        }
    }


    for(i = 0; i < matrix_dimensions_a[0]; i++){
        for(j = 0; j < matrix_dimensions_b[1]; j++){
            for(k = 0; k < matrix_dimensions_b[0]; k++){
                output_matrix[(i*matrix_dimensions_b[1]) + j] += matrix_a[(i*matrix_dimensions_a[1]) + j] * matrix_b[(i*matrix_dimensions_b[1]) + j];
            }
        }
    }

}
//
//void get_mmio_dimensions(FILE *f, int* dimensions_out){
//    //Dimensions will be placed in the dimensions_out parameter with the following form:
//    // dimensions_out[0] = rows
//    // dimensions_out[1] = columns
//    // dimensions_out[2] = number of non-zero coordinates
//    MM_typecode matcode;
//    int rows, columns, num_non_zero;
//
//    if (mm_read_banner(f, &matcode) != 0)
//    {
//        printf("Could not read banner for input file.\n");
//        exit(1);
//    }
//
//    if(!mm_is_coordinate(matcode)){
//        printf("This application only supports MMIO files in the coordinate format");
//        exit(1);
//    }
//
//    mm_read_mtx_crd_size(f, &rows, &columns, &num_non_zero);
//
//    dimensions_out[0] = rows ;
//    dimensions_out[1] = columns ;
//    dimensions_out[2] = num_non_zero;
//
//}

void create_matrix_array(FILE *f, int * dimensions, float *matrix_out){
    //subtract 1 from the demi

    int columns = dimensions[1];
    int num_non_zero = dimensions[2];
    int *row_index;
    int *column_index;
    double *val;

    row_index = (int *) malloc(num_non_zero * sizeof(int));
    column_index = (int *) malloc(num_non_zero * sizeof(int));
    val = (double *) malloc(num_non_zero * sizeof(double));

    for(int i = 0; i < num_non_zero; i++){
        fscanf(f, "%d %d %lg\n", &row_index[i], &column_index[i], &val[i]);
        row_index[i]--;
        column_index[i]--;
        matrix_out[(row_index[i] * columns) + column_index[i]] = val[i];
    }

}

void write_matrix_to_file(FILE *f, int dimensions[], float matrix_out[]){
    MM_typecode matout_code;

    mm_initialize_typecode(&matout_code);
    mm_set_matrix(&matout_code);
    mm_set_coordinate(&matout_code);
    mm_set_real(&matout_code);

    mm_write_banner(f, matout_code);
    mm_write_mtx_crd_size(f, dimensions[0], dimensions[1], dimensions[2]);

    for(int i=0; i < dimensions[0]; i ++){
        for (int j = 0; j < dimensions[1]; j++){
            fprintf(f, "%d %d %g\n", i+1, j+1, matrix_out[coord_to_index(i, j, dimensions[1])]);
        }
    }
}


int main(int argc, char *argv[]) {

    //MM_typecode matcode;
    FILE *input_matrix_file_a;
    FILE *input_matrix_file_b;
    FILE *output_file;
    int matrix_dimensions_a[3];
    int matrix_dimensions_b[3];
    float *matrix_a;
    float *matrix_b;
    float *output_matrix;
    int n = 10;
//    if(argc != 4){
//        printf("Usage is: matrixmatrix matrix_a_file matrix_b_file output_file");
//    }
//    if ((input_matrix_file_a = fopen(argv[1], "r")) == NULL) {
//        printf("Failed to open input matrix_a file\n");
//        exit(1);
//    }
    // Fetch the dimensions for the first matrix_a
    //get_mmio_dimensions(input_matrix_file_a, matrix_dimensions_a);

    //allocate memory for first matrix_a
    //int matrix_a[matrix_dimensions_a[0]*matrix_dimensions_a[1]] = {0};
    hipMallocManaged(&matrix_a,n*n * sizeof(float));

    //Populate the first matrix_a
    //create_matrix_array(input_matrix_file_a, matrix_dimensions_a, matrix_a);
    random_square_matrix(n, matrix_a);

    //fclose(input_matrix_file_a);


//    if ((input_matrix_file_b = fopen(argv[2], "r")) == NULL) {
//        printf("Failed to open input matrix_b file\n");
//        exit(1);
//    }
    // Fetch the dimensions for the matrix_b
   // get_mmio_dimensions(input_matrix_file_b, matrix_dimensions_b);

    //allocate memory for matrix_b
    hipMallocManaged(&matrix_b,n *n * sizeof(float));

    //Populate the matrix_b
    //create_matrix_array(input_matrix_file_b, matrix_dimensions_b, matrix_b);
    random_square_matrix(n, matrix_b);

    //fclose(input_matrix_file_b);

    hipMallocManaged(&output_matrix, n*n * sizeof(float));
    dim3 gridDim = 1;
    dim3 blockDim = 1000;
    matrix_matrix_multiply<<<gridDim,blockDim>>>(matrix_dimensions_a, matrix_dimensions_b, matrix_a, matrix_b, output_matrix);

    // for(int i=0; i < matrix_dimensions_a[0]; i ++){
    //     for (int j = 0; j < matrix_dimensions_b[1]; j++){
    //         printf("%g\t", output_matrix[coord_to_index(i, j, matrix_dimensions_b[1])]);
    //     }
    //     printf("\n");
    // }
    hipDeviceSynchronize();
    int output_dimensions[3] = {matrix_dimensions_a[0], matrix_dimensions_b[1], matrix_dimensions_a[0] * matrix_dimensions_b[1] };

    //Open file to write out
    if ((output_file = fopen(argv[3], "w")) == NULL) {
        printf("Failed to open output matrix file\n");
        exit(1);
    }

    write_matrix_to_file(output_file, output_dimensions, output_matrix);
    //fclose(output_file);
    //Release memory
    //free(matrix_a);
    //free(matrix_b);
    return(0);
}
